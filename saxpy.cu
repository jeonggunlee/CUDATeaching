#include <stdio.h>
#include <hip/hip_runtime.h>
// CUDA Saxpy 
__global__ void saxpy(int n, float a, float* x, float* y)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    y[idx] = a*x[idx] + y[idx];
}

int main()
{
    int n = 1024*256;
    float a = 0.15;
    float x[n];
    float y[n];
    float h_y[n];
    float* d_x;
    float* d_y;

    // initialize
    for(int i=0; i<n; i++) {
        y[i] = i*0.01 + 0.4;
        h_y[i] = i*0.01 + 0.4;
        x[i] = i*0.02 + 0.2;
    }

    // cudaMalloc ( void** devPtr, size_t size ) 
    hipMalloc(&d_x, n*sizeof(float));
    hipMalloc(&d_y, n*sizeof(float));

    // __host__​cudaError_t cudaMemcpy ( void* dst, const void* src, size_t count, cudaMemcpyKind kind )
    hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);

    int threadPerBlock = 256;
    //int blocks = n / 256;
    // int n, float a, float x, float y
    saxpy<<<n / 256, threadPerBlock>>>(n, a, d_x, d_y);

    hipDeviceSynchronize();
    hipMemcpy(y, d_y, n*sizeof(float), hipMemcpyDeviceToHost);

    // Correct Function
    for(int i=0; i<n; i++) h_y[i] = a*x[i] + h_y[i];

    // Verification
    float diff;
    float errorSum=0;
    for(int i=0; i<n; i++) {
        diff = y[i] - h_y[i];
        errorSum = errorSum + diff*diff;
    }
    printf("Error Sum = %f\n", errorSum);
    for(int i=0; i<10; i++) {
        printf("Host Results:Device Results %f %f\n", h_y[i], y[i]);
    }

    // __host__ ​ __device__ ​cudaError_t cudaFree ( void* devPtr ) 
    hipFree(d_x);
    hipFree(d_y);
}